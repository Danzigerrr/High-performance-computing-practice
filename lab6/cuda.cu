#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>
#include "numgen.c"

// Function to print error message and exit
__host__
void errorexit(const char *s) {
    printf("\n%s", s);	
    exit(EXIT_FAILURE);	 	
}

__device__
int isPrime(long value) {
    if (value < 2) return 0;          
    if (value == 2) return 1;         
    if (value % 2 == 0) return 0;     

    long sqroot = sqrt((double)value);
    for (long i = 3; i <= sqroot; i += 2) {
        if (value % i == 0) {
            return 0;                 
        }
    }
    return 1; // Number is prime
}

// Kernel to count primes in a given range
__global__
void countPrimes(const unsigned long int *numbers, int *result, long inputArgument) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < inputArgument) {
        result[idx] = isPrime(numbers[idx]);   // Store result of prime check in result array
    }
}

int main(int argc, char **argv) {
    Args ins__args;
    parseArgs(&ins__args, &argc, argv);       
  
    long inputArgument = ins__args.arg;       
    unsigned long int *numbers = (unsigned long int *)malloc(inputArgument * sizeof(unsigned long int));
    numgen(inputArgument, numbers);           

    int threadsinblock = 1024;             
    int blocksingrid = (inputArgument + threadsinblock - 1) / threadsinblock;  // Calculate number of blocks in grid
    
    // Allocate memory on the host
    int *h_results = (int *)malloc(inputArgument * sizeof(int));
    if (!h_results) errorexit("Error allocating memory on the host");

    // Allocate memory on the GPU
    unsigned long int *d_numbers = NULL;
    int *d_results = NULL;
      // memory for input number
    if (hipSuccess != hipMalloc((void **)&d_numbers, inputArgument * sizeof(unsigned long int)))
        errorexit("Error allocating memory on the GPU for numbers");
      // memory for results
    if (hipSuccess != hipMalloc((void **)&d_results, inputArgument * sizeof(int)))
        errorexit("Error allocating memory on the GPU for results");

    // Copy data from host to device
    if (hipSuccess != hipMemcpy(d_numbers, numbers, inputArgument * sizeof(unsigned long int), hipMemcpyHostToDevice))
        errorexit("Error copying numbers to GPU");

    // count excetion time - begin
    struct timeval ins__tstart, ins__tstop;
    gettimeofday(&ins__tstart, NULL);

    // Launch the kernel to count primes
    countPrimes<<<blocksingrid, threadsinblock>>>(d_numbers, d_results, inputArgument);
    if (hipSuccess != hipGetLastError())
        errorexit("Error during kernel launch");

    // Copy results from GPU to host
    if (hipSuccess != hipMemcpy(h_results, d_results, inputArgument * sizeof(int), hipMemcpyDeviceToHost))
        errorexit("Error copying results from GPU");

    // On the host - Sum the results
    long long result = 0;
    for (long i = 0; i < inputArgument; i++) {
        result += h_results[i];
    }

    printf("\nThe final result is %lld\n", result);

    // Free allocated memory
    free(numbers);
    free(h_results);
    if (hipSuccess != hipFree(d_numbers))
        errorexit("Error when deallocating space on the GPU for numbers");
    if (hipSuccess != hipFree(d_results))
        errorexit("Error when deallocating space on the GPU for results");


    // Stop timing the kernel execution
    gettimeofday(&ins__tstop, NULL);
    ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);
    
    return 0;
}
